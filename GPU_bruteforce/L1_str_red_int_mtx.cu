
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<math.h>
#include<stdlib.h>
#define length 4096
#define NUM_OF_THREADS 16384
#define NUM_OF_BITS 8 * sizeof(unsigned long long int)

__global__ void func(int* d_mtx_to_vec, unsigned long long int steps, unsigned long long int steps_remainder, int *d_L1_vector, int *d_L1_strategy, int iLonger, int iShorter){
	int i, l;

	int temp[length], vect[NUM_OF_BITS + 1], product = 0, L1 = 0;
	unsigned long long int number, index, iMax, iMin, logical;

	index = blockIdx.x * blockDim.x + threadIdx.x;
	iMax = (index + 1) *(steps) - 1;
	iMin = index * (steps);
	if(index < (steps_remainder) ) iMax += index + 1;
	else iMax += (steps_remainder);
	if(index <= (steps_remainder)) iMin += index;
	else iMin += (steps_remainder);
//	if(index == 1) printf("index: %llu, iMin: %llu, iMax: %llu\n", index, iMin, iMax);
		 
		 for(number = iMin; number <= iMax; number++){
		 product = 0;
		 for(l=0; l < iLonger; l++) {temp[l] = d_mtx_to_vec[(iShorter-1) * iLonger + l];}
			for(i = 0 ; (iShorter - 1) > i; i++){
				logical = number & ((unsigned long long int) 1 << i);
				logical = logical == 0 ? 0 : 1;
				vect[i] = (int) 2 * logical - 1;
				for(l=0; l < iLonger; l++){
					temp[l] += d_mtx_to_vec[i * iLonger + l] * vect[i];
				}
			}
			for(l= 0; l < iLonger; l++) {product += abs(temp[l]);}
			if(product > L1){
				L1 = product;
				for(l=0; l<(iShorter - 1); l++){d_L1_strategy[index * (iShorter - 1) + l] = vect[l];}
			}
		}

d_L1_vector[index] = L1;
}

int** mtx_read(int *iRows, int *iCols, char* fileName){
	int i = 0,j = 0, k = 0;
	int *row, **mtx, value;
	
	mtx = NULL;
	row = NULL;
	
	char g, cNum[256];
	
	FILE *fp;
	fp = fopen(fileName,"r");
	
	do{
		
		g = fgetc(fp);
		
//		
		if((((g - '0') < 10) && ((g - '0') >= 0)) || (g == 'e') || ( g == 'E') || (g == '.') || (g == '+') || (g == '-')) {cNum[i] = g; i++;}
		else {
			cNum[i] = '\0'; 
			if(cNum[0] != '\0') {sscanf(cNum, "%d", &value); j++; i = 0;  row = (int*) realloc(row, j * sizeof(int)); row[j-1] = value;}
			if( ((g == '\n') || (g == EOF)) && (j > 0)){ *iCols = j; j = 0; k++; mtx = (int**) realloc(mtx, k * sizeof(int*)); mtx[k-1] = row; row = NULL;}
		}

	}while(!feof(fp));
	*iRows = k;
printf("rows: %d, cols: %d\n",*iRows, *iCols); 
	fclose(fp);
return mtx;
}

void fileN(char *fileName, char** argv, int *argc){
	int r;
	FILE *fp;
	if((*argc) < 2) {
		do{
			printf("Please give me a filename: "); 
			r = scanf("%s",fileName);
		}while(r != 1);
	}
	else sprintf(fileName,"%s", argv[1]);

	fp = fopen(fileName, "r");
	if(fp == NULL) {
		do{
			printf("Please give me a filename that exist within this directory: ");
			r = scanf("%s",fileName);
			fp = fopen(fileName, "r");
		}while(fp == NULL);
	}
	fclose(fp);
}

int main(int argc, char *argv[]){
hipDeviceProp_t devProp;
hipGetDeviceProperties(&devProp, 0);
     char fileName[1024];
     fileN(fileName, argv, &argc);     
     int i, j, iMax, iRows, iCols, **mtx, *mtx_to_vec, *d_mtx_to_vec, iShorter, iLonger;

     mtx = mtx_read(&iRows, &iCols, fileName);
     mtx_to_vec = (int*)calloc(iRows * iCols, sizeof(int));
	if(iRows > iCols){
		for(j = 0; j < iCols; j++){
			for(i = 0; i < iRows; i++){
				mtx_to_vec[j * iRows + i] = mtx[i][j];
			}
		}
	}
	else{
		for(i = 0; i < iRows; i++){
			for(j = 0; j < iCols; j++){
				mtx_to_vec[i * iCols + j] = mtx[i][j];
			}
		}
	}

	if(iRows < iCols) {iShorter = iRows; iLonger = iCols;}
	else {iShorter = iCols; iLonger = iRows;}
	if(iShorter > (NUM_OF_BITS + 1)) {printf("Matrix is too big. The number of rows or columns can not be more than %lu.\n", NUM_OF_BITS + 1); return 0;}
	if(iLonger > length) {printf("Matrix is too big. The length variable %d should be bigger or equal than %d.\n", length, iLonger); return 0;}
	hipMalloc((void**)&d_mtx_to_vec, iRows * iCols * sizeof(int));
     	hipMalloc((void**)&d_mtx_to_vec, iRows * iCols * sizeof(int));
     	unsigned long long int steps, steps_remainder, Inner_num = (unsigned long long int) 1 << (iShorter - 1), copyNum;
	copyNum = NUM_OF_THREADS > Inner_num ? Inner_num : NUM_OF_THREADS;
	int *L1_vector, *d_L1_vector, L1_max = 0, *L1_strategy, *d_L1_strategy, num_ofBlock = (int) ceil((float)copyNum/(float)devProp.warpSize), num_ofThread = copyNum < devProp.warpSize ? copyNum : devProp.warpSize;

	steps=Inner_num/copyNum; steps_remainder = Inner_num % copyNum;

	L1_vector = (int*) malloc(copyNum * sizeof(int));
	L1_strategy = (int*) malloc(copyNum * (iShorter - 1) * sizeof(int));

	hipMalloc((void**)&d_L1_vector, copyNum * sizeof(int));
	hipMalloc((void**)&d_L1_strategy, copyNum * (iShorter - 1) * sizeof(int));
	
	hipMemcpy(d_mtx_to_vec, mtx_to_vec, iRows * iCols * sizeof(int), hipMemcpyHostToDevice);
printf("num_ofBlock: %d, num_ofThread: %d\n",num_ofBlock,num_ofThread);
	 func<<<num_ofBlock,num_ofThread>>>(d_mtx_to_vec, steps, steps_remainder, d_L1_vector, d_L1_strategy, iLonger, iShorter);
	hipMemcpy(L1_vector, d_L1_vector, copyNum * sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(L1_strategy, d_L1_strategy, copyNum * (iShorter - 1) * sizeof(int), hipMemcpyDeviceToHost);
	 
	for(i = 0; i < copyNum; i++){
		if(L1_max < L1_vector[i]) {L1_max = L1_vector[i]; iMax = i;}
	}
FILE *fp;
fp = fopen("strategy_L1.txt", "w");
	for(i=0; i<(iShorter - 1); i++) {fprintf(fp, "%d\n", L1_strategy[iMax * (iShorter - 1) + i]);}
	fprintf(fp,"1\n");
fclose(fp);

	printf("L1 is: %d\n",L1_max);

	free(L1_vector);
	free(L1_strategy);
	free(mtx_to_vec);

	hipFree(d_L1_vector);
	hipFree(d_L1_strategy);
	hipFree(d_mtx_to_vec);

     return 0;     
}
