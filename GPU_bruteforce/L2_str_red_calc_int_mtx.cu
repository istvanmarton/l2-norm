
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<math.h>
#include<stdlib.h>
#define length 4096
#define NUM_OF_THREADS 16384
#define NUM_OF_BITS 8 * sizeof(unsigned long long int)

__global__ void func(int* d_mtx_to_vec, unsigned long long int steps, unsigned long long int steps_remainder, int *d_L2_vector, int *d_L2_strategy, int iRows, int iCols){
	int i, l;
	int temp_negative[length], temp_positive[length], vect[NUM_OF_BITS], product, L2;
	unsigned long long int number, index, iMax, iMin, iNumofZeros, iNum_temp;

	int logical;
	index = blockIdx.x * blockDim.x + threadIdx.x;

	iMax = (index + 1) *(steps) - 1;
	iMin = index * (steps);
	if(index < (steps_remainder) ) iMax += index + 1;
	else iMax += (steps_remainder);
	if(index <= (steps_remainder)) iMin += index;
	else iMin += (steps_remainder);
//	if(index == 0) printf("index: %llu, iMin: %llu, iMax: %llu\n", index, iMin, iMax);
		 number = iMin;
		 for(l=0; l < iCols; l++) {temp_negative[l] = 0; temp_positive[l] = d_mtx_to_vec[(iRows-1) * iCols + l];}
		 product = 0;
			for(i = 0 ; (iRows - 1) > i; i++){
				iNumofZeros=(unsigned long long int) 1 << i;
				iNum_temp = (unsigned long long int) iNumofZeros << 1;
				logical = ((number+ iNumofZeros)/iNum_temp) % 2;
				vect[i] = (int) 2 * logical - 1;
//if(index == 2) printf("%d, ",vect[i]);
					if(vect[i] > 0){for(l=0; l < iCols; l++){temp_positive[l] += d_mtx_to_vec[i * iCols + l]; }}
					else {for(l=0; l < iCols; l++){temp_negative[l] += d_mtx_to_vec[i * iCols + l]; }}				
			}
			for(l= 0; l < iCols; l++) {product += abs(temp_negative[l]) + abs(temp_positive[l]);}
			L2 = product;
			for(l=0; l<(iRows - 1); l++){d_L2_strategy[index * (iRows - 1) + l] = vect[l];}

     for(number=iMin + 1; number <= iMax; number++){
//if(index == 2) printf("\n");
		 product = 0;
			for(i = 0 ; (iRows - 1) > i; i++){
				iNumofZeros=(unsigned long long int) 1 << i;
				iNum_temp = (unsigned long long int) iNumofZeros << 1;
				if( ((number+ iNumofZeros) % iNum_temp) ==0 ) {vect[i]=-vect[i] ;					
					if(vect[i] > 0){for(l=0; l < iCols; l++){temp_positive[l] += d_mtx_to_vec[i * iCols + l]; temp_negative[l] -= d_mtx_to_vec[i * iCols + l]; }}
					else {for(l=0; l < iCols; l++){temp_positive[l] -= d_mtx_to_vec[i * iCols + l]; temp_negative[l] += d_mtx_to_vec[i * iCols + l]; }}
				break;
				}
//if(index == 2) printf("%d, ",vect[i]);
            		}
	     for(l = 0; l < (iCols ); l++) {product += abs(temp_negative[l]) + abs(temp_positive[l]);}
	     if(product > L2) {L2 = product;
		for(l=0; l<(iRows - 1); l++){d_L2_strategy[index * (iRows - 1) + l] = vect[l];}
		}
     }
d_L2_vector[index] = L2;
}

int** mtx_read(int *iRows, int *iCols, char* fileName){
	int i = 0,j = 0, k = 0;
	int *row, **mtx, value;
	
	mtx = NULL;
	row = NULL;
	
	char g, cNum[256];
	
	FILE *fp;
	fp = fopen(fileName,"r");
	
	do{
		g = fgetc(fp);	
		if((((g - '0') < 10) && ((g - '0') >= 0)) || (g == 'e') || ( g == 'E') || (g == '.') || (g == '+') || (g == '-')) {cNum[i] = g; i++;}
		else {
			cNum[i] = '\0'; 
			if(cNum[0] != '\0') {sscanf(cNum, "%d", &value); j++; i = 0;  row = (int*) realloc(row, j * sizeof(int)); row[j-1] = value;}
			if( ((g == '\n') || (g == EOF)) && (j > 0)){*iCols = j; j = 0; k++; mtx = (int**) realloc(mtx, k * sizeof(int*)); mtx[k-1] = row; row = NULL;}
		}
		
	}while(!feof(fp));
	*iRows = k;
printf("rows: %d, cols: %d\n",*iRows, *iCols); 
	fclose(fp);
return mtx;
}

void fileN(char *fileName, char** argv, int *argc){
	int r;
	FILE *fp;
	if((*argc) < 2) {
		do{
			printf("Please give me a filename: "); 
			r = scanf("%s",fileName);
		}while(r != 1);
	}
	else sprintf(fileName,"%s", argv[1]);

	fp = fopen(fileName, "r");
	if(fp == NULL) {
		do{
			printf("Please give me a filename that exist within this directory: ");
			r = scanf("%s",fileName);
			fp = fopen(fileName, "r");
		}while(fp == NULL);
	}
	fclose(fp);
}

int main(int argc, char *argv[]){
hipDeviceProp_t devProp;
hipGetDeviceProperties(&devProp, 0);
     char fileName[1024];
     fileN(fileName, argv, &argc);     
     int i, j, iMax, iRows, iCols, **mtx, *mtx_to_vec, *d_mtx_to_vec;

     mtx = mtx_read(&iRows, &iCols, fileName);
     mtx_to_vec = (int*)calloc(iRows * iCols, sizeof(int));

		for(i = 0; i < iRows; i++){
			for(j = 0; j < iCols; j++){
				mtx_to_vec[i * iCols + j] = mtx[i][j];
			}
		}

	if(iRows > (NUM_OF_BITS)) {printf("Matrix is too big. The number of rows or columns can not be more than %lu.\n", NUM_OF_BITS); return 0;}
	if(iCols > length) {printf("Matrix is too big. The length variable %d should be bigger or equal than %d.\n", length, iCols); return 0;}
	hipMalloc((void**)&d_mtx_to_vec, iRows * iCols * sizeof(int));
     	hipMalloc((void**)&d_mtx_to_vec, iRows * iCols * sizeof(int));
     	unsigned long long int steps, steps_remainder, Inner_num = (unsigned long long int) 1 << (iRows - 1), copyNum;
	copyNum = NUM_OF_THREADS > Inner_num ? Inner_num : NUM_OF_THREADS;
	int *L2_vector, *d_L2_vector, L2_max = 0, *L2_strategy, *d_L2_strategy, num_ofBlock = (int) ceil((float)copyNum/(float)devProp.warpSize), num_ofThread = copyNum < devProp.warpSize ? copyNum : devProp.warpSize;

	steps=Inner_num/copyNum; steps_remainder = Inner_num % copyNum;

	L2_vector = (int*) malloc(copyNum * sizeof(int));
	L2_strategy = (int*) malloc(copyNum * (iRows - 1) * sizeof(int));

	hipMalloc((void**)&d_L2_vector, copyNum * sizeof(int));
	hipMalloc((void**)&d_L2_strategy, copyNum * (iRows - 1) * sizeof(int));
	
	hipMemcpy(d_mtx_to_vec, mtx_to_vec, iRows * iCols * sizeof(int), hipMemcpyHostToDevice);
printf("num_ofBlock: %d, num_ofThread: %d\n",num_ofBlock,num_ofThread);
	 func<<<num_ofBlock,num_ofThread>>>(d_mtx_to_vec, steps, steps_remainder, d_L2_vector, d_L2_strategy, iRows, iCols);
	hipMemcpy(L2_vector, d_L2_vector, copyNum * sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(L2_strategy, d_L2_strategy, copyNum * (iRows - 1) * sizeof(int), hipMemcpyDeviceToHost);
	 
	for(i = 0; i < copyNum; i++){
		if(L2_max < L2_vector[i]) {L2_max = L2_vector[i]; iMax = i;}
	}
FILE *fp;
fp = fopen("strategy_L2.txt", "w");
	for(i=0; i<(iRows - 1); i++) {fprintf(fp, "%d\n", L2_strategy[iMax * (iRows - 1) + i]);}
	fprintf(fp,"1\n");
fclose(fp);

	printf("L2 is: %d\n",L2_max);

	free(L2_vector);
	free(L2_strategy);
	free(mtx_to_vec);

	hipFree(d_L2_vector);
	hipFree(d_L2_strategy);
	hipFree(d_mtx_to_vec);

     return 0;     
}
