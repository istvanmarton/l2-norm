
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<math.h>
#include<stdlib.h>
#define length 128

__global__ void func(int* d_mtx_to_vec, unsigned long long int numThreads, unsigned long long int steps, unsigned long long int steps_remainder, int *d_L2_vector, int *d_L2_strategy, int iLonger, int iShorter){
	int i, l;
//printf("Itt meg jol vagyok.\n");

//	int *temp = new int[iLonger], product = 0 ,L1 = 0, *vect = new int[iShorter], elojel;
	int temp_L2_negative[length], temp_L2_positive[length], vect[length], product = 0, elojel, L2 = 0;
	unsigned long long int number, index, iMax, iMin, iNumofZeros, iNum_temp;
	int logical;
	index = blockIdx.x * blockDim.x + threadIdx.x;

//int iiiMax = 0;
	
	iMax = (index + 1) *(steps) - 1;
	iMin = index * (steps);
	if(index < (steps_remainder) ) iMax += index + 1;
	else iMax += (steps_remainder);
	if(index <= (steps_remainder)) iMin += index;
	else iMin += (steps_remainder);
//	printf("index: %llu, iMin: %llu, iMax: %llu\n", index, iMin, iMax);
		 number = iMin;
		 for(l=0; l < iLonger; l++) {temp_L2_negative[l] = 0; temp_L2_positive[l] = 0;}
		 product = 0;
			for(i = (iShorter - 1); i >=0 ; i--){
				iNum_temp = 1 << (i+1);
				iNumofZeros=iNum_temp>>1;
				logical = ((number+ iNumofZeros)/iNum_temp) % 2;
				logical = logical == 0 ? 0 : 1;
//				printf("\nhj: %u, j: %d\n", h[j], j); 
 //             printf("i: %d, j: %d\n", i,j);
//				printf("%d", logical == 0 ? 0 : 1);
				vect[i] = (int) 2 * logical - 1;
				for(l=0; l < iLonger; l++){
					if(vect[i] < 0) { temp_L2_negative[l] -= d_mtx_to_vec[i * iLonger + l];}
					else {		  temp_L2_positive[l] += d_mtx_to_vec[i * iLonger + l];}
				}
			}
			for(l= 0; l < iLonger; l++) {product += abs(temp_L2_positive[l]) + abs(temp_L2_negative[l]);}
			L2 = product;
			for(l=0; l<(iShorter); l++){d_L2_strategy[index * (iShorter) + l] = vect[l];}

     for(number=iMin + 1; number <= iMax; number++){
//		 printf("k: %llu\n",number);
		 product = 0; //szamlalo=0;

			for(i = (iShorter - 1); i >=0 ; i--){
				iNum_temp = 1 << (i+1);
				iNumofZeros=iNum_temp>>1;
				logical = ((number+ iNumofZeros)/iNum_temp) % 2;
				if(vect[i] != (2*logical - 1)) {elojel = (int) 2*logical - 1; 
					for(l=0; l < iLonger; l++){
						if(elojel < 0){
							temp_L2_positive[l] -= 2 * d_mtx_to_vec[i * iLonger + l];
							temp_L2_negative[l] += 2 * d_mtx_to_vec[i * iLonger + l];
						}
						else{
							temp_L2_positive[l] += 2 * d_mtx_to_vec[i * iLonger + l];
							temp_L2_negative[l] -= 2 * d_mtx_to_vec[i * iLonger + l];	
						}

					}
//				szamlalo++;
//				if(szamlalo > 1) printf("A szamlalo: %d\n", szamlalo);
				vect[i] = (int) 2 * logical - 1;
				break;
			}
								
//				printf("vector: %d\n", vect[i]);
//              printf("logical: %d, mask: %d\n", logical, mask << i);
            		}
	     for(l = 0; l < (iLonger ); l++) {product += abs(temp_L2_positive[l]) + abs(temp_L2_negative[l]);}
	     if(product > L2) {L2 = product;
		for(l=0; l<(iShorter); l++){d_L2_strategy[index * (iShorter) + l] = vect[l]; /*printf("%d\n", vect[k]);*/}
		}
//		 meret++;
//      printf("\n");
     }
d_L2_vector[index] = L2;
//printf("%max: %llu\n",iiiMax);
}

int** mtx_read(int *iRows, int *iCols, char* fileName){
printf("%s\n",fileName);
	int i = 0,j = 0, k = 0;
	int *row, **mtx, value;
	
	mtx = NULL; //(double**)malloc(sizeof(double*));
	row = NULL;
//	mtx[0] = row;
	
	char g, cNum[256];
	
	FILE *fp;
	fp = fopen(fileName,"r");
	
	do{
		
		g = fgetc(fp); 
		
//		
		if((((g - '0') < 10) && ((g - '0') >= 0)) || (g == 'e') || ( g == 'E') || (g == '.') || (g == '+') || (g == '-')) {cNum[i] = g;  /*printf("i: %d\tc: %c\n", i, cNum[i]);*/ i++;}
		else {
//			printf("%d\t%d\n",i,j);
			cNum[i] = '\0'; 
			if(cNum[0] != '\0') {i = 0; j++; sscanf(cNum, "%d", &value); /*printf("value: %d, col: %d, row: %d,\n", value, i,j);*/ row = (int*) realloc(row, j * sizeof(int)); row[j-1] = value; 
			if((g == '\n') || (g == EOF)){/*printf("j: %d\n",j);*/ *iCols = j; j = 0; k++; /*printf("k: %d\n",k);*/ mtx = (int**) realloc(mtx, k * sizeof(int*)); mtx[k-1] = row; /*printf("mtx: %d\t%d\t%d\n", mtx[k-1][0], mtx[k-1][1], mtx[k-1][2]);*/ /*free(row);*/ row = NULL;}
			}
		}
		
		
			
//		cNum[i] =  ? ;
		
	}while(!feof(fp));
//	printf("j: %d\n",j);
	*iRows = k;
printf("rows: %d, cols: %d\n",*iRows, *iCols); 
	fclose(fp);
//	free(row);
return mtx;
}

void fileN(char *fileName, char** argv, int *argc){
	if((*argc) < 2) {printf("Please give me a filename: "); scanf("%s",fileName);}
	else sprintf(fileName,"%s", argv[1]);//fileName = argv[1];
//	printf("fileName :%s", fileName);
}


int main(int argc, char *argv[]){
     char fileName[1024];
     fileN(fileName, argv, &argc);     
     printf("%s\n",fileName);
     int i, j, iMax, iRows, iCols, **mtx, *mtx_to_vec, *d_mtx_to_vec;
     mtx = mtx_read(&iRows, &iCols, fileName);
     mtx_to_vec = (int*)calloc(iRows * iCols, sizeof(int));

		for(i = 0; i < iRows; i++){
			for(j = 0; j < iCols; j++){
				mtx_to_vec[i * iCols + j] = mtx[i][j];
			}
		}

	hipMalloc((void**)&d_mtx_to_vec, iRows * iCols * sizeof(int));
//for(i=0; i < iRows*iCols; i++) {printf("%d\n",mtx_to_vec[i]);}
     hipMalloc((void**)&d_mtx_to_vec, iRows * iCols * sizeof(int));
     unsigned long long int numThreads, steps, steps_remainder, Inner_num = (unsigned long long int) 1 << (iRows), copyNum;
	 
//	 printf("numThread: %d\n", numThread);

	 numThreads = (unsigned long long int) 10752; steps=Inner_num/numThreads; steps_remainder = Inner_num % numThreads;// ( (int) Inner_num/(numThreads));

	int *L2_vector, *d_L2_vector, L2_max = 0, *L2_strategy, *d_L2_strategy;

	copyNum = numThreads > Inner_num ? Inner_num : numThreads;
	printf("copyNum: %llu\n", copyNum);
	L2_vector = (int*) malloc(copyNum * sizeof(int));
	L2_strategy = (int*) malloc(copyNum * (iRows) * sizeof(int));

	hipMalloc((void**)&d_L2_vector, copyNum * sizeof(int));
	hipMalloc((void**)&d_L2_strategy, copyNum * (iRows) * sizeof(int));
	
	hipMemcpy(d_mtx_to_vec, mtx_to_vec, iRows * iCols * sizeof(int), hipMemcpyHostToDevice);
printf("NUM of Blocks: %llu\n", (unsigned long long int) ceil((int) numThreads/32));
	 func<<<(unsigned long long int) ceil((int) numThreads/32),32>>>(d_mtx_to_vec, numThreads, steps, steps_remainder, d_L2_vector, d_L2_strategy, iCols, iRows);
	hipMemcpy(L2_vector, d_L2_vector, copyNum * sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(L2_strategy, d_L2_strategy, copyNum * (iRows) * sizeof(int), hipMemcpyDeviceToHost);
	 
	for(i = 0; i < copyNum; i++){
		if(L2_max < L2_vector[i]) {L2_max = L2_vector[i]; iMax = i;}
	}
FILE *fp;
fp = fopen("strategy_L2.txt", "w");	
	printf("The strategy is:\n");
	for(i=0; i<(iRows); i++) {printf("%d\n", L2_strategy[iMax * (iRows) + i]); fprintf(fp, "%d\n", L2_strategy[iMax * (iRows) + i]);}
fclose(fp);

//     printf("szam: %d\n", h >> 1);
//     printf("szam: %d\n", h >> 1);
	printf("L2 is: %d\n",L2_max);

	free(L2_vector);
	free(L2_strategy);
	free(mtx_to_vec);

	hipFree(d_L2_vector);
	hipFree(d_L2_strategy);
	hipFree(d_mtx_to_vec);

     return 0;     
}
