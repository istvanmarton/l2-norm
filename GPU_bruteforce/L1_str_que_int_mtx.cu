
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<math.h>
#include<stdlib.h>
#define length 128

__global__ void func(int* d_mtx_to_vec, unsigned long long int numThreads, unsigned long long int steps, unsigned long long int steps_remainder, int *d_L1_vector, int *d_L1_strategy, int iLonger, int iShorter){
	int i, l;
//printf("Itt meg jol vagyok.\n");

//	int *temp = new int[iLonger], product = 0 ,L1 = 0, *vect = new int[iShorter], elojel;
	int temp[length], vect[length], product = 0, elojel, L1 = 0;
	unsigned long long int number, index, iMax, iMin, iNumofZeros, iNum_temp;
	int logical;
	index = blockIdx.x * blockDim.x + threadIdx.x;

//int iiiMax = 0;
	
	iMax = (index + 1) *(steps) - 1;
	iMin = index * (steps);
	if(index < (steps_remainder) ) iMax += index + 1;
	else iMax += (steps_remainder);
	if(index <= (steps_remainder)) iMin += index;
	else iMin += (steps_remainder);
//	printf("index: %llu, iMin: %llu, iMax: %llu\n", index, iMin, iMax);
		 number = iMin;
		 for(l=0; l < iLonger; l++) {temp[l] = d_mtx_to_vec[(iShorter) * iLonger + l];}
		 product = 0;
			for(i = (iShorter - 1); i >=0 ; i--){
				iNum_temp = 1 << (i+1);
				iNumofZeros=iNum_temp>>1;
				logical = ((number+ iNumofZeros)/iNum_temp) % 2;
				logical = logical == 0 ? 0 : 1;
//				printf("\nhj: %u, j: %d\n", h[j], j); 
 //             printf("i: %d, j: %d\n", i,j);
//				printf("%d", logical == 0 ? 0 : 1);
				vect[i] = (int) 2 * logical - 1;
				for(l=0; l < iLonger; l++){
					temp[l] += d_mtx_to_vec[i * iLonger + l] * vect[i];
//					printf("mtx[%d][%d]: %d, vect: %d, temp: %d\n", i, l, vect[i], d_mtx_to_vec[i * iLonger + l] * vect[i]);
				}
			}
			for(l= 0; l < iLonger; l++) {product += abs(temp[l]);}
			L1 = product;
			for(l=0; l<(iShorter); l++){d_L1_strategy[index * (iShorter) + l] = vect[l];}

     for(number=iMin + 1; number <= iMax; number++){
//		 printf("k: %llu\n",number);
		 product = 0; //szamlalo=0;

			for(i = (iShorter - 1); i >=0 ; i--){
				iNum_temp = 1 << (i+1);
				iNumofZeros=iNum_temp>>1;
				logical = ((number+ iNumofZeros)/iNum_temp) % 2;
				if(vect[i] != (2*logical - 1)) {elojel = (int) 2*logical - 1; 
					for(l=0; l < iLonger; l++){temp[l] += 2 * d_mtx_to_vec[i * iLonger + l] * elojel; /*if(iiiMax < i) iiiMax = i; printf("i: %llu, elojel: %d\n",i, elojel);*/}
//				szamlalo++;
//				if(szamlalo > 1) printf("A szamlalo: %d\n", szamlalo);
				vect[i] = (int) 2 * logical - 1;
				break;
			}
								
//				printf("vector: %d\n", vect[i]);
//              printf("logical: %d, mask: %d\n", logical, mask << i);
            		}
	     for(l = 0; l < (iLonger ); l++) {product += abs(temp[l]);}
	     if(product > L1) {L1 = product;
		for(l=0; l<(iShorter); l++){d_L1_strategy[index * (iShorter) + l] = vect[l]; /*printf("%d\n", vect[k]);*/}
		}
//		 meret++;
//      printf("\n");
     }
//printf("L1: %d, meret: %d\n", L1, meret);
//printf("L1: %d\n", L1);
d_L1_vector[index] = L1;
//printf("%max: %llu\n",iiiMax);
}

int** mtx_read(int *iRows, int *iCols, char* fileName){
printf("%s\n",fileName);
	int i = 0,j = 0, k = 0;
	int *row, **mtx, value;
	
	mtx = NULL; //(double**)malloc(sizeof(double*));
	row = NULL;
//	mtx[0] = row;
	
	char g, cNum[256];
	
	FILE *fp;
	fp = fopen(fileName,"r");
	
	do{
		
		g = fgetc(fp); 
		
//		
		if((((g - '0') < 10) && ((g - '0') >= 0)) || (g == 'e') || ( g == 'E') || (g == '.') || (g == '+') || (g == '-')) {cNum[i] = g;  /*printf("i: %d\tc: %c\n", i, cNum[i]);*/ i++;}
		else {
//			printf("%d\t%d\n",i,j);
			cNum[i] = '\0'; 
			if(cNum[0] != '\0') {i = 0; j++; sscanf(cNum, "%d", &value); /*printf("value: %d, col: %d, row: %d,\n", value, i,j);*/ row = (int*) realloc(row, j * sizeof(int)); row[j-1] = value; 
			if((g == '\n') || (g == EOF)){/*printf("j: %d\n",j);*/ *iCols = j; j = 0; k++; /*printf("k: %d\n",k);*/ mtx = (int**) realloc(mtx, k * sizeof(int*)); mtx[k-1] = row; /*printf("mtx: %d\t%d\t%d\n", mtx[k-1][0], mtx[k-1][1], mtx[k-1][2]);*/ /*free(row);*/ row = NULL;}
			}
		}
		
		
			
//		cNum[i] =  ? ;
		
	}while(!feof(fp));
//	printf("j: %d\n",j);
	*iRows = k;
printf("rows: %d, cols: %d\n",*iRows, *iCols); 
	fclose(fp);
//	free(row);
return mtx;
}

void fileN(char *fileName, char** argv, int *argc){
	if((*argc) < 2) {printf("Please give me a filename: "); scanf("%s",fileName);}
	else sprintf(fileName,"%s", argv[1]);//fileName = argv[1];
//	printf("fileName :%s", fileName);
}


int main(int argc, char *argv[]){
     char fileName[1024];
     fileN(fileName, argv, &argc);     
     printf("%s\n",fileName);
     int i, j, iMax, iRows, iCols, **mtx, *mtx_to_vec, *d_mtx_to_vec, iShorter, iLonger;
     mtx = mtx_read(&iRows, &iCols, fileName);
     mtx_to_vec = (int*)calloc(iRows * iCols, sizeof(int));
	if(iRows > iCols){
		for(j = 0; j < iCols; j++){
			for(i = 0; i < iRows; i++){
				mtx_to_vec[j * iRows + i] = mtx[i][j];
			}
		}
	}
	else{
		for(i = 0; i < iRows; i++){
			for(j = 0; j < iCols; j++){
				mtx_to_vec[i * iCols + j] = mtx[i][j];
			}
		}
	}

	if(iRows < iCols) {iShorter = iRows; iLonger = iCols;}
	else {iShorter = iCols; iLonger = iRows;}
//printf("host, ishorter: %d, iLonger: %d\n", iShorter, iLonger);
	hipMalloc((void**)&d_mtx_to_vec, iRows * iCols * sizeof(int));
//for(i=0; i < iRows*iCols; i++) {printf("%d\n",mtx_to_vec[i]);}
     hipMalloc((void**)&d_mtx_to_vec, iRows * iCols * sizeof(int));
     unsigned long long int numThreads, steps, steps_remainder, Inner_num = (unsigned long long int) 1 << (iShorter), copyNum;
	 
//	 printf("numThread: %d\n", numThread);

	 numThreads = (unsigned long long int) 10752; steps=Inner_num/numThreads; steps_remainder = Inner_num % numThreads;// ( (int) Inner_num/(numThreads));

	int *L1_vector, *d_L1_vector, L1_max = 0, *L1_strategy, *d_L1_strategy;

	copyNum = numThreads > Inner_num ? Inner_num : numThreads;
	printf("copyNum: %llu\n", copyNum);
	L1_vector = (int*) malloc(copyNum * sizeof(int));
	L1_strategy = (int*) malloc(copyNum * (iShorter) * sizeof(int));

	hipMalloc((void**)&d_L1_vector, copyNum * sizeof(int));
	hipMalloc((void**)&d_L1_strategy, copyNum * (iShorter) * sizeof(int));
	
	hipMemcpy(d_mtx_to_vec, mtx_to_vec, iRows * iCols * sizeof(int), hipMemcpyHostToDevice);
printf("NUM of Blocks: %llu\n", (unsigned long long int) ceil((int) numThreads/32));
	 func<<<(unsigned long long int) ceil((int) numThreads/32),32>>>(d_mtx_to_vec, numThreads, steps, steps_remainder, d_L1_vector, d_L1_strategy, iLonger, iShorter);
	hipMemcpy(L1_vector, d_L1_vector, copyNum * sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(L1_strategy, d_L1_strategy, copyNum * (iShorter) * sizeof(int), hipMemcpyDeviceToHost);
	 
	for(i = 0; i < copyNum; i++){
		if(L1_max < L1_vector[i]) {L1_max = L1_vector[i]; iMax = i;}
	}
FILE *fp;
fp = fopen("strategy.txt", "w");	
	printf("The strategy is:\n");
	for(i=0; i<(iShorter); i++) {printf("%d\n", L1_strategy[iMax * (iShorter) + i]); fprintf(fp, "%d\n", L1_strategy[iMax * (iShorter) + i]);}
fclose(fp);

//     printf("szam: %d\n", h >> 1);
//     printf("szam: %d\n", h >> 1);
	printf("L1 is: %d\n",L1_max);

	free(L1_vector);
	free(L1_strategy);
	free(mtx_to_vec);

	hipFree(d_L1_vector);
	hipFree(d_L1_strategy);
	hipFree(d_mtx_to_vec);

     return 0;     
}
