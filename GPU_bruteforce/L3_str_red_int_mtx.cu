
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<math.h>
#include<stdlib.h>
#define length 4096
#define NUM_OF_THREADS 10752
#define NUM_OF_BITS 8 * sizeof(unsigned long long int)

__global__ void func(int* d_mtx_to_vec, unsigned long long int steps, unsigned long long int steps_remainder, int *d_L3_vector, int *d_L3_strategy, int iRows, int iCols, int n, unsigned long long int *d_iNumPower){
	int i, l;
	int temp_0[length], temp_1[length], temp_2[length], vect[NUM_OF_BITS + 1], product, L3 = 0;
	unsigned long long int number, index, iMax, iMin, iNum_temp;

	int logical;
	index = blockIdx.x * blockDim.x + threadIdx.x;

	iMax = (index + 1) *(steps) - 1;
	iMin = index * (steps);
	if(index < (steps_remainder) ) iMax += index + 1;
	else iMax += (steps_remainder);
	if(index <= (steps_remainder)) iMin += index;
	else iMin += (steps_remainder);
//	if(index == 0) printf("index: %llu, iMin: %llu, iMax: %llu\n", index, iMin, iMax);

	for(number=iMin; number <= iMax; number++){
		 for(l=0; l < iCols; l++) {temp_0[l] = d_mtx_to_vec[(iRows-1) * iCols + l]; temp_1[l] = 0; temp_2[l] = 0;}
		 product = 0;
			for(i = 0 ; (iRows - 1) > i; i++){
				iNum_temp = d_iNumPower[i];//pow(n,i+1);
				logical = (number/iNum_temp) % n; //printf("%d, ", logical);
				vect[i] = logical;
					switch(logical)
					{
					case 0:
						for(l=0; l < iCols; l++){temp_0[l] += d_mtx_to_vec[i * iCols + l]; }
					break;
					case 1:
						for(l=0; l < iCols; l++){temp_1[l] += d_mtx_to_vec[i * iCols + l]; }
					break;
					case 2:
						for(l=0; l < iCols; l++){temp_2[l] += d_mtx_to_vec[i * iCols + l]; }
					break;
					}				
			}
			//printf("\n");
			for(l= 0; l < iCols; l++) {product += abs(temp_0[l]) + abs(temp_1[l]) + abs(temp_2[l]);}
			if(product > L3) {
				L3 = product;
				for(l=0; l<(iRows - 1); l++){d_L3_strategy[index * (iRows - 1) + l] = vect[l];}
			}
	}
d_L3_vector[index] = L3;
}

int** mtx_read(int *iRows, int *iCols, char* fileName){
	int i = 0,j = 0, k = 0;
	int *row, **mtx, value;
	
	mtx = NULL;
	row = NULL;
	
	char g, cNum[256];
	
	FILE *fp;
	fp = fopen(fileName,"r");
	
	do{
		g = fgetc(fp);	
		if((((g - '0') < 10) && ((g - '0') >= 0)) || (g == 'e') || ( g == 'E') || (g == '.') || (g == '+') || (g == '-')) {cNum[i] = g; i++;}
		else {
			cNum[i] = '\0'; 
			if(cNum[0] != '\0') {sscanf(cNum, "%d", &value); j++; i = 0;  row = (int*) realloc(row, j * sizeof(int)); row[j-1] = value;}
			if( ((g == '\n') || (g == EOF)) && (j > 0)){*iCols = j; j = 0; k++; mtx = (int**) realloc(mtx, k * sizeof(int*)); mtx[k-1] = row; row = NULL;}
		}
		
	}while(!feof(fp));
	*iRows = k;
printf("rows: %d, cols: %d\n",*iRows, *iCols); 
	fclose(fp);
return mtx;
}

void fileN(char *fileName, char** argv, int *argc){
	int r;
	FILE *fp;
	if((*argc) < 2) {
		do{
			printf("Please give me a filename: "); 
			r = scanf("%s",fileName);
		}while(r != 1);
	}
	else sprintf(fileName,"%s", argv[1]);

	fp = fopen(fileName, "r");
	if(fp == NULL) {
		do{
			printf("Please give me a filename that exist within this directory: ");
			r = scanf("%s",fileName);
			fp = fopen(fileName, "r");
		}while(fp == NULL);
	}
	fclose(fp);
}

int main(int argc, char *argv[]){
hipDeviceProp_t devProp;
hipGetDeviceProperties(&devProp, 0);
     char fileName[1024];
     fileN(fileName, argv, &argc);     
     int i, j, iMax, iRows, iCols, **mtx, *mtx_to_vec, *d_mtx_to_vec, n = 3, maxRows = (int) (floor (NUM_OF_BITS / log2(n)) + 1);
     printf("NUM_OF_BITS: %lu, maxRows: %d\n", NUM_OF_BITS ,maxRows);
     mtx = mtx_read(&iRows, &iCols, fileName);
     mtx_to_vec = (int*)calloc(iRows * iCols, sizeof(int));

		for(i = 0; i < iRows; i++){
			for(j = 0; j < iCols; j++){
				mtx_to_vec[i * iCols + j] = mtx[i][j];
			}
		}
	if( iRows > maxRows) {printf("Matrix is too big. The number of rows can not be more than %d.\n", maxRows); return 0;}
	if(iCols > length) {printf("Matrix is too big. The length variable %d should be bigger or equal than %d.\n", length, iCols); return 0;}
	hipMalloc((void**)&d_mtx_to_vec, iRows * iCols * sizeof(int));
     	hipMalloc((void**)&d_mtx_to_vec, iRows * iCols * sizeof(int));
     	unsigned long long int steps, steps_remainder, Inner_num = (unsigned long long int) pow(3, iRows - 1)/*1 << (iShorter - 1)*/, copyNum, *iNumPower, *d_iNumPower;
	copyNum = NUM_OF_THREADS > Inner_num ? Inner_num : NUM_OF_THREADS;
	int *L3_vector, *d_L3_vector, L3_max = 0, *L3_strategy, *d_L3_strategy, num_ofBlock = (int) ceil((float)copyNum/(float)devProp.warpSize), num_ofThread = copyNum < devProp.warpSize ? copyNum : devProp.warpSize;

	steps=Inner_num/copyNum; steps_remainder = Inner_num % copyNum;

	L3_vector = (int*) malloc(copyNum * sizeof(int));
	L3_strategy = (int*) malloc(copyNum * (iRows - 1) * sizeof(int));
	iNumPower = (unsigned long long int*) malloc(maxRows * sizeof(unsigned long long int));

	hipMalloc((void**)&d_L3_vector, copyNum * sizeof(int));
	hipMalloc((void**)&d_L3_strategy, copyNum * (iRows - 1) * sizeof(int));
	hipMalloc((void**)&d_iNumPower, maxRows * sizeof(unsigned long long int));
     	for(i = 0; i < maxRows; i++){iNumPower[i] = pow(n, i);} //printf("%llu\n", iNumPower[maxRows-1]);
	hipMemcpy(d_iNumPower, iNumPower, maxRows * sizeof(unsigned long long int), hipMemcpyHostToDevice);
	hipMemcpy(d_mtx_to_vec, mtx_to_vec, iRows * iCols * sizeof(int), hipMemcpyHostToDevice);
printf("num_ofBlock: %d, num_ofThread: %d\n",num_ofBlock,num_ofThread);
	 func<<<num_ofBlock,num_ofThread>>>(d_mtx_to_vec, steps, steps_remainder, d_L3_vector, d_L3_strategy, iRows, iCols, n, d_iNumPower);
	hipMemcpy(L3_vector, d_L3_vector, copyNum * sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(L3_strategy, d_L3_strategy, copyNum * (iRows - 1) * sizeof(int), hipMemcpyDeviceToHost);	 
	for(i = 0; i < copyNum; i++){
		if(L3_max < L3_vector[i]) {L3_max = L3_vector[i]; iMax = i;}
	}
FILE *fp;
fp = fopen("strategy_L3.txt", "w");
	for(i=0; i<(iRows - 1); i++) {fprintf(fp, "%d\n", L3_strategy[iMax * (iRows - 1) + i]);}
	fprintf(fp,"0\n");
fclose(fp);

	printf("L3 is: %d\n",L3_max);

	free(L3_vector);
	free(L3_strategy);
	free(mtx_to_vec);
	free(iNumPower);

	hipFree(d_L3_vector);
	hipFree(d_L3_strategy);
	hipFree(d_mtx_to_vec);
	hipFree(d_iNumPower);

     return 0;     
}
