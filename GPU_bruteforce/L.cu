/*****************************

WRITTEN BY ISTVÁN MÁRTON

*****************************/


#include <hip/hip_runtime.h>
#include<stdio.h>
#include<math.h>
#include<stdlib.h>
#define length 4096
#define NUM_OF_THREADS 16384
#define RANK_OF_NORM 20
#define NUM_OF_BITS 8 * sizeof(unsigned long long int)

__global__ void L1(int* d_mtx_to_vec, unsigned long long int steps, unsigned long long int steps_remainder, int *d_L1_vector, int *d_L1_strategy, int iLonger, int iShorter){ // This function calculates the L1 norm on the GPU.
	int i, l, logical;
	int temp[length], vect[NUM_OF_BITS], product, L1;
	unsigned long long int number, index, iMax, iMin, iNumofZeros, iNum_temp;

	index = blockIdx.x * blockDim.x + threadIdx.x; // Index of threads.

	iMax = (index + 1) *(steps) - 1; // This part calculates the minimal (iMin-th) and the maximal (iMax-th) word of the binary reflected Gray code for which the calculations must be performed by a given thread.
	iMin = index * (steps);
	if(index < (steps_remainder) ) iMax += index + 1;
	else iMax += (steps_remainder);
	if(index <= (steps_remainder)) iMin += index;
	else iMin += (steps_remainder);
	number = iMin;

	for(l=0; l < iLonger; l++) {temp[l] = d_mtx_to_vec[l];} // As the code can consider a row of the matrix with fixed sign, it considers the first row of the matrix with +1.
	product = 0;
	for(i = 1 ; iShorter > i; i++){
		iNum_temp = (unsigned long long int) 1 << i; // iNum_temp and iNumofZeros are coefficients to determine the number-th word of BRGC.
		iNumofZeros=(unsigned long long int) iNum_temp >> 1;		
		logical = ((number+ iNumofZeros)/iNum_temp) % 2; // logical can be 0 and 1. logical is the number-th word and i-th digit of the BRGC.
		vect[i] = (int) 2 * logical - 1; // vect is the possible strategy vector. It's elements consists of +1 and -1.
			if(vect[i] > 0){for(l=0; l < iLonger; l++){temp[l] += d_mtx_to_vec[i * iLonger + l]; }} // The code determines the vector-matrix multiplication belonging to the number-th word of the BRGC.
			else {for(l=0; l < iLonger; l++){temp[l] -= d_mtx_to_vec[i * iLonger + l]; }}				
	}
	for(l= 0; l < iLonger; l++) {product += abs(temp[l]);} // The code calculates the L1 value belonging to the number-th word of the BRGC.
	L1 = product; 
	for(l=1; l<iShorter; l++){d_L1_strategy[index * (iShorter - 1) + l - 1] = vect[l];} // The program stores the strategy vector belonging to the number-th BRGC word in the d_L1_strategy vector.

	for(number=iMin + 1; number <= iMax; number++){ //The code determines the BRGC words till number variable reaches iMax.
		product = 0;
		for(i = 1 ; iShorter > i; i++){
			iNum_temp = (unsigned long long int) 1 << i; // iNum_temp and iNumofZeros are coefficients to determine the j-th word of BRGC.
			iNumofZeros=(unsigned long long int) iNum_temp >> 1;
			if( ((number+ iNumofZeros) % iNum_temp) == 0 ) {vect[i]=-vect[i]; // The code calculates if there is a change in the i-th digit in the BRGC.
				if(vect[i] > 0){for(l=0; l < iLonger; l++){temp[l] += 2 * d_mtx_to_vec[i * iLonger + l]; }} // When the i-th digit is changed, the code changes the result of the vector-matrix multiplication. It only deals with the i-th row of the matrix.
				else {for(l=0; l < iLonger; l++){temp[l] -= 2 *d_mtx_to_vec[i * iLonger + l]; }}
			break;
		}
            		}
	for(l = 0; l < (iLonger ); l++) {product += abs(temp[l]);} // Calculates the number-th possible L1 value.
		if(product > L1) {L1 = product; // If the current possible L1 value is greater than the previous, it modifies both the value and the corresponding strategy vector as well.
			for(l=1; l<iShorter; l++){d_L1_strategy[index * (iShorter - 1) + l - 1] = vect[l];}
		}
    }
d_L1_vector[index] = L1; // Every thread writes the biggest found L1 value to the d_L1_vector. This vector will be copied to the host memory.
}

__global__ void L2(int* d_mtx_to_vec, unsigned long long int steps, unsigned long long int steps_remainder, int *d_L2_vector, int *d_L2_strategy, int iRows, int iCols){
	int i, l, logical;
	int temp_negative[length], temp_positive[length], vect[NUM_OF_BITS], product, L2;
	unsigned long long int number, index, iMax, iMin, iNumofZeros, iNum_temp;

	index = blockIdx.x * blockDim.x + threadIdx.x;

	iMax = (index + 1) *(steps) - 1;
	iMin = index * (steps);
	if(index < (steps_remainder) ) iMax += index + 1;
	else iMax += (steps_remainder);
	if(index <= (steps_remainder)) iMin += index;
	else iMin += (steps_remainder);
	number = iMin;

	for(l=0; l < iCols; l++) {temp_negative[l] = 0; temp_positive[l] = d_mtx_to_vec[l]; }
	product = 0;
	for(i = 1 ; iRows > i; i++){
		iNum_temp = (unsigned long long int) 1 << i;
		iNumofZeros=(unsigned long long int) iNum_temp >> 1;		
		logical = ((number+ iNumofZeros)/iNum_temp) % 2;
		vect[i] = (int) 2 * logical - 1;
			if(vect[i] > 0){for(l=0; l < iCols; l++){temp_positive[l] += d_mtx_to_vec[i * iCols + l]; }}
			else {for(l=0; l < iCols; l++){temp_negative[l] += d_mtx_to_vec[i * iCols + l]; }}				
	}

	for(l= 0; l < iCols; l++) {product += abs(temp_negative[l]) + abs(temp_positive[l]);}
	L2 = product;
	for(l=1; l<iRows; l++){d_L2_strategy[index * (iRows - 1) + l - 1] = vect[l];}

	for(number=iMin + 1; number <= iMax; number++){
	product = 0;
	for(i = 1 ; iRows > i; i++){
		iNum_temp = (unsigned long long int) 1 << i;
		iNumofZeros=(unsigned long long int) iNum_temp >> 1;
		if( ((number+ iNumofZeros) % iNum_temp) == 0 ) {vect[i]=-vect[i] ;				
			if(vect[i] > 0){for(l=0; l < iCols; l++){temp_positive[l] += d_mtx_to_vec[i * iCols + l]; temp_negative[l] -= d_mtx_to_vec[i * iCols + l]; }}
			else {for(l=0; l < iCols; l++){temp_positive[l] -= d_mtx_to_vec[i * iCols + l]; temp_negative[l] += d_mtx_to_vec[i * iCols + l]; }}
			break;
		}
	}
	
	for(l = 0; l < (iCols ); l++) {product += abs(temp_negative[l]) + abs(temp_positive[l]);}
	if(product > L2) {L2 = product;
		for(l=1; l<iRows; l++){d_L2_strategy[index * (iRows - 1) + l - 1] = vect[l];}
	}
    }
d_L2_vector[index] = L2;
}

__global__ void L3(int* d_mtx_to_vec, unsigned long long int steps, unsigned long long int steps_remainder, int *d_L3_vector, int *d_L3_strategy, int iRows, int iCols, unsigned long long int *d_iNumPower){
	int i, l, helper[6] = {0, 1, 2, 2, 1, 0};

	int temp_0[length], temp_1[length], temp_2[length], vect[NUM_OF_BITS + 1], product, L3 = 0, logical, temporary;
	unsigned long long int number, index, iMax, iMin, divide;

	index = blockIdx.x * blockDim.x + threadIdx.x;

	iMax = (index + 1) *(steps) - 1;
	iMin = index * (steps);
	if(index < (steps_remainder) ) iMax += index + 1;
	else iMax += (steps_remainder);
	if(index <= (steps_remainder)) iMin += index;
	else iMin += (steps_remainder);
	
	number = iMin;
	for(l=0; l < iCols; l++) {temp_0[l] = d_mtx_to_vec[(iRows-1) * iCols + l]; temp_1[l] = 0; temp_2[l] = 0;}
	product = 0;
	for(i = 0 ; (iRows - 1) > i; i++){
		logical = (number/d_iNumPower[i]) % 6; // Determines the ternary reflected Gray code (TRGC). d_iNumPower is a vector consisting of the power of 3. This vector was copied from the host to speed up the calculation of TRGC as the power of 3 does not need to be determined every time.
		vect[i] = helper[logical];
		switch(vect[i])
			{
			case 0:
			for(l=0; l < iCols; l++){temp_0[l] += d_mtx_to_vec[i * iCols + l]; }
			break;
			case 1:
			for(l=0; l < iCols; l++){temp_1[l] += d_mtx_to_vec[i * iCols + l]; }
			break;
			case 2:
			for(l=0; l < iCols; l++){temp_2[l] += d_mtx_to_vec[i * iCols + l]; }
			break;
			}				
	}
	
	for(l= 0; l < iCols; l++) {product += abs(temp_0[l]) + abs(temp_1[l]) + abs(temp_2[l]);}
	L3 = product;
	for(l=0; l<(iRows - 1); l++){d_L3_strategy[index * (iRows - 1) + l] = vect[l];}

	for(number=iMin + 1; number <= iMax; number++){
      	product = 0;
		for(i = 0 ; (iRows - 1) > i; i++){
			divide = number/d_iNumPower[i];
			logical = divide % 3; //printf("%d, ", logical);
			if(logical) {
				logical = divide % 6;
				temporary = helper[logical];
				if( (vect[i] == 0)  && (temporary == 1) ) {for(l=0; l < iCols; l++){temp_0[l] -= d_mtx_to_vec[i * iCols + l]; temp_1[l] += d_mtx_to_vec[i * iCols + l];}}
					else if((vect[i] == 1)  && (temporary == 2)) {for(l=0; l < iCols; l++){temp_1[l] -= d_mtx_to_vec[i * iCols + l]; temp_2[l] += d_mtx_to_vec[i * iCols + l];}}
					else if((vect[i] == 2)  && (temporary == 1)) {for(l=0; l < iCols; l++){temp_1[l] += d_mtx_to_vec[i * iCols + l]; temp_2[l] -= d_mtx_to_vec[i * iCols + l];}}
					else {for(l=0; l < iCols; l++){temp_0[l] += d_mtx_to_vec[i * iCols + l]; temp_1[l] -= d_mtx_to_vec[i * iCols + l];}}
					vect[i] = temporary;
					break;
			}				
		}
		
	for(l= 0; l < iCols; l++) {product += abs(temp_0[l]) + abs(temp_1[l]) + abs(temp_2[l]);}
		if(product > L3) {
			L3 = product;
			for(l=0; l<(iRows - 1); l++){d_L3_strategy[index * (iRows - 1) + l] = vect[l];}
		}
	}
d_L3_vector[index] = L3;
}

__global__ void Ln(int* d_mtx_to_vec, int* d_iHelper, unsigned long long int steps, unsigned long long int steps_remainder, int *d_Ln_vector, int *d_Ln_strategy, int iRows, int iCols, int n, unsigned long long int *d_iNumPower){
	int i, l;
	int temp[RANK_OF_NORM][length], vect[NUM_OF_BITS + 1], product, Ln = 0;
	unsigned long long int number, index, iMax, iMin, divide;

	int logical, temporary;
	index = blockIdx.x * blockDim.x + threadIdx.x;
	iMax = (index + 1) *(steps) - 1;
	iMin = index * (steps);
	if(index < (steps_remainder) ) iMax += index + 1;
	else iMax += (steps_remainder);
	if(index <= (steps_remainder)) iMin += index;
	else iMin += (steps_remainder);
	number = iMin;

	for(l=0; l < iCols; l++) {
		temp[0][l] = d_mtx_to_vec[(iRows-1) * iCols + l]; 
		for(i=1; i<n; i++){
			temp[i][l] = 0;
		}
	}
	
	product = 0;
	for(i = 0 ; (iRows - 1) > i; i++){
		logical = (number/d_iNumPower[i]) % (2*n); 
		vect[i] = d_iHelper[logical]; //d_iHelper is a vector consisting of the power of n. It helps to determine the words of the n-ary Gray code.
		for(l=0; l < iCols; l++){temp[vect[i]][l] += d_mtx_to_vec[i * iCols + l]; }				
	}

	for(l= 0; l < iCols; l++) {
		for(i=0; i < n; i++){
			product += abs(temp[i][l]);
		}
	}
	Ln = product;
	for(l=0; l<(iRows - 1); l++){d_Ln_strategy[index * (iRows - 1) + l] = vect[l];}

	for(number=iMin + 1; number <= iMax; number++){
		product = 0;
		for(i = 0 ; (iRows - 1) > i; i++){
			divide = number/d_iNumPower[i];
			logical = divide % n; //printf("%d, ", logical);
			if(logical) {
				logical = divide % (2*n);
				temporary = d_iHelper[logical];
				for(l=0; l < iCols; l++) {temp[vect[i]][l] -= d_mtx_to_vec[i * iCols + l]; temp[temporary][l] += d_mtx_to_vec[i * iCols + l];}
				vect[i] = temporary;
				break;
			}			
		}
		for(l= 0; l < iCols; l++) {
			for(i=0; i < n; i++){
				product += abs(temp[i][l]);
			}
		}
		if(product > Ln) {
			Ln = product;
			for(l=0; l<(iRows - 1); l++){d_Ln_strategy[index * (iRows - 1) + l] = vect[l];}
		}
	}
d_Ln_vector[index] = Ln;
}

int** mtx_read(int *iRows, int *iCols, char* fileName){
	int i = 0,j = 0, k = 0;
	int *row, **mtx, value;
	
	mtx = NULL;
	row = NULL;
	
	char g, cNum[256];
	
	FILE *fp;
	fp = fopen(fileName,"r");
	
	do{
		g = fgetc(fp);	
		if((((g - '0') < 10) && ((g - '0') >= 0)) || (g == 'e') || ( g == 'E') || (g == '.') || (g == '+') || (g == '-')) {cNum[i] = g; i++;}
		else {
			cNum[i] = '\0'; 
			if(cNum[0] != '\0') {sscanf(cNum, "%d", &value); j++; i = 0;  row = (int*) realloc(row, j * sizeof(int)); row[j-1] = value;}
			if( ((g == '\n') || (g == EOF)) && (j > 0)){*iCols = j; j = 0; k++; mtx = (int**) realloc(mtx, k * sizeof(int*)); mtx[k-1] = row; row = NULL;}
		}
		
	}while(!feof(fp));
	*iRows = k;
printf("rows: %d, cols: %d\n",*iRows, *iCols); 
	fclose(fp);
return mtx;
}

void fileN(char *fileName, char** argv, int *argc){
	int r;
	FILE *fp;
	fp = NULL;
	if((*argc) < 2) {
		while(fp == NULL){
			printf("Please give me a filename that exist within this directory: ");
			r = scanf("%s",fileName);
			if(r == 0) {printf("Something went wrong when a filename was typed!\n");}
			fp = fopen(fileName, "r");
		}
	}
	else if((*argc) < 3){
		sprintf(fileName,"%s", argv[1]);
		fp = fopen(fileName, "r");
		while(fp == NULL){
			printf("Please give me a filename that exist within this directory: ");
			r = scanf("%s",fileName);
			if(r == 0) {printf("Something went wrong when a filename was typed!\n");}
			fp = fopen(fileName, "r");
		}
	}
	else if((*argc) < 4){
		sprintf(fileName,"%s", argv[1]);
		fp = fopen(fileName, "r");
		if(fp == NULL) {
			sprintf(fileName,"%s", argv[2]);
			fp = fopen(fileName, "r");
			while(fp == NULL){
				printf("Please give me a filename that exist within this directory: ");
				r = scanf("%s",fileName);
				if(r == 0) {printf("Something went wrong when a filename was typed!\n");}
				fp = fopen(fileName, "r");
			}
		}
	}
	else {
		printf("Too many parameters!\n");
		exit(-1);
	}
	fclose(fp);
}

void nNumber(int* p, char** argv, int *argc){
	int sd, n = 0;
	char line[256];
	
	if((*argc) < 2) {
		printf("Please give an integer bigger than zero: ");
		fgets(line, sizeof(line), stdin);
		sd = sscanf(line, "%d", &n);
		while((sd == 0) || (n < 1)){
			if(sd == 0) {printf("The entry must be an integer! Please give an integer bigger than zero: ");}
			if(n < 1  && sd != 0) {printf("Please give an integer bigger than zero: ");}
			fgets(line, sizeof(line), stdin);
			sd = sscanf(line, "%d", &n);
			//printf("sd value: %lu\n", sizeof(line));
		}
	}
	else if((*argc) < 3){
		sd = sscanf(argv[1], "%d", &n);
		while((sd == 0) || (n < 1)){
			if(sd == 0) {printf("Please give an integer bigger than zero: ");}
			if(n < 1  && sd != 0) {printf("Please give an integer bigger than zero: ");}
			fgets(line, sizeof(line), stdin);
			sd = sscanf(line, "%d", &n);
			//printf("sd value: %lu\n", sizeof(line));
		}
	}
	else if((*argc) < 4){
		sd = sscanf(argv[1], "%d", &n);
		if((sd == 0) || (n < 1)) {sd = sscanf(argv[2], "%d", &n);}
		while((sd == 0) || (n < 1)){
			if(sd == 0) {printf("Please give an integer bigger than zero: ");}
			if(n < 1  && sd != 0) {printf("Please give an integer bigger than zero: ");}
			fgets(line, sizeof(line), stdin);
			sd = sscanf(line, "%d", &n);
			//printf("sd value: %lu\n", sizeof(line));
		}
	}
	else {
		printf("Too many parameters!\n");
		exit(-1);
	}

	if(n > RANK_OF_NORM) {printf("The order of the L norm is too big. Please increase the RANK_OF_NORM variable in the code to %d and compile and run it again!\n", n); exit(-1);}
	*p = n;
}

void calc_Lnorm(int* n, int* iRows, int* iCols, int** mtx){
	FILE *fp;
	char fileOutput[1024]; // The variable 'fileOutput' is the name of the file, the 
	int i, j, iMax, *mtx_to_vec, *d_mtx_to_vec, maxRows, *Ln_vector, *d_Ln_vector, Ln_max, *Ln_strategy, *d_Ln_strategy, num_ofBlock, num_ofThread; //i and j are the indices of the input matrix; iMax is the variable belonging to the strategy vector, the index of the strategy vector; mtx_to_vec: the input matrix is converted to a vector in the host; d_mtx_to_vec: the converted matrix in the device; maxRows: them maximal number of rows (in case of L1, the maximal number of rows or columns) of the matrix the program can deal with, this number is determined by the order of the L norm that should be calculated; Ln_vector and d_Ln_vector are the two vectors containing the possible L norms belonging to a given thread in the host and device respectively; Ln_max: The maximal possible value of the L norm in the host; Ln_strategy and d_Ln_strategy are the vector containing all the possible strategy vectors belonging to different threads; num_ofBlock is the number of blocks the program uses; num_ofThread is the number of thread in a block
	unsigned long long int steps, steps_remainder, Inner_num, copyNum; //Inner_num: the number of possible L value the device should calculate; copyNum: the number of possible L values that will be copied from the device to the host 
	hipDeviceProp_t devProp; // devProp contains the number of cores a warp contain.
	hipGetDeviceProperties(&devProp, 0);
	mtx_to_vec = (int*)calloc(*iRows * *iCols, sizeof(int)); // allocating memory for the mtx_to_vec variable

	if(NUM_OF_THREADS < 1) {printf("The  NUM_OF_THREADS variable must be greater than 1. Modify it, and compile again!\n"); exit(-1);}
	if(*n == 1){ // if the order of the L norm is 1 then this part of the code will be executed.
		int iShorter, iLonger; //iShorter is the number of rows or columns, whichever is less; iLonger is the number of rows or columns, whichever is bigger
		if( *iRows > *iCols ){ //In this if else sequence the code transposes the matrix if necessary and transform a matrix into a vector
			for(j = 0; j < *iCols; j++){
				for(i = 0; i < *iRows; i++){
					mtx_to_vec[j * *iRows + i] = mtx[i][j];
				}
			}
		}
		else{
			for(i = 0; i < *iRows; i++){
				for(j = 0; j < *iCols; j++){
					mtx_to_vec[i * *iCols + j] = mtx[i][j];
				}
			}
		}
		if(*iRows < *iCols) {iShorter = *iRows; iLonger = *iCols;}
		else {iShorter = *iCols; iLonger = *iRows;}
		if(iShorter > (NUM_OF_BITS)) {printf("Matrix is too big. The number of rows or columns can not be more than %lu.\n", NUM_OF_BITS); exit(-1);}
		if(iLonger > length) {printf("Matrix is too big. The length variable %d should be bigger or equal than %d.\n", length, iLonger); exit(-1);}
		hipMalloc((void**)&d_mtx_to_vec, iShorter * iLonger * sizeof(int)); // Allocating memory for the matrix in the device.
		Inner_num = (unsigned long long int) 1 << (iShorter - 1); //The number of possible L values that the device should calculate is 2^(iShorter-1)
		copyNum = NUM_OF_THREADS > Inner_num ? Inner_num : NUM_OF_THREADS; // The possible number of L norms can not be more than the number of threads
		num_ofThread = copyNum < devProp.warpSize ? copyNum : devProp.warpSize; // Number of threads in a block can not be bigger than the number of warps.
		num_ofBlock = copyNum/num_ofThread; copyNum = num_ofBlock * num_ofThread; //The number of blocks the code uses.
		steps = Inner_num/copyNum; steps_remainder = Inner_num % copyNum;
		Ln_vector = (int*) malloc(copyNum * sizeof(int)); // The code allocates memory in the host for the possible L norms.
		Ln_strategy = (int*) malloc(copyNum * (iShorter - 1) * sizeof(int)); // The code allocates memory for the possible strategies belonging to L norms in the host.
		hipMalloc((void**)&d_Ln_vector, copyNum * sizeof(int)); // The code allocates memory in the device for the possible L norms.
		hipMalloc((void**)&d_Ln_strategy, copyNum * (iShorter - 1) * sizeof(int)); // The code allocates memory for the possible strategies belonging to L norms in the device.
		hipMemcpy(d_mtx_to_vec, mtx_to_vec, iShorter * iLonger * sizeof(int), hipMemcpyHostToDevice); // The matrix is copied from RAM to GPU memory.
		printf("num_ofBlock: %d, num_ofThread: %d\n",num_ofBlock,num_ofThread);
		L1<<<num_ofBlock,num_ofThread>>>(d_mtx_to_vec, steps, steps_remainder, d_Ln_vector, d_Ln_strategy, iLonger, iShorter); // The calculation of the L1 norm with GPU.
		hipMemcpy(Ln_vector, d_Ln_vector, copyNum * sizeof(int), hipMemcpyDeviceToHost); // Copy the possible L norm values from device to host.
		hipMemcpy(Ln_strategy, d_Ln_strategy, copyNum * (iShorter - 1) * sizeof(int), hipMemcpyDeviceToHost); // Copy the possible strategies belonging to L norm values from device to host.
		Ln_max = Ln_vector[0]; iMax = 0; // Determining the maximal element of the Ln_vector which is the L norm, and the index of the strategy vector as well.
		for(i = 1; i < copyNum; i++){ if(Ln_max < Ln_vector[i]) {Ln_max = Ln_vector[i]; iMax = i;}}

		fp = fopen("strategy_L1.txt", "w");// Print out the strategy vector to file.
		fprintf(fp,"1\n");
		for(i=0; i<(iShorter - 1); i++) {fprintf(fp, "%d\n", Ln_strategy[iMax * (iShorter - 1) + i]);}
		fclose(fp);
	}
	else if(*n == 2){ // if the order of the L norm is 2 then this part of the code will be executed.

			for(i = 0; i < *iRows; i++){
				for(j = 0; j < *iCols; j++){
					mtx_to_vec[i * *iCols + j] = mtx[i][j];
				}
			}

		if(*iRows > (NUM_OF_BITS)) {printf("Matrix is too big. The number of rows or columns can not be more than %lu.\n", NUM_OF_BITS); exit(-1);}
		if(*iCols > length) {printf("Matrix is too big. The length variable %d should be bigger or equal than %d.\n", length, *iCols); exit(-1);}
		hipMalloc((void**)&d_mtx_to_vec, *iRows * *iCols * sizeof(int));
		Inner_num = (unsigned long long int) 1 << (*iRows - 1);
		copyNum = NUM_OF_THREADS > Inner_num ? Inner_num : NUM_OF_THREADS; // The possible number of L norms can not be more than the number of threads
		num_ofThread = copyNum < devProp.warpSize ? copyNum : devProp.warpSize; // Number of threads in a block can not be bigger than the number of warps.
		num_ofBlock = copyNum/num_ofThread; copyNum = num_ofBlock * num_ofThread; //The number of blocks the code uses.
		steps=Inner_num/copyNum; steps_remainder = Inner_num % copyNum;
		Ln_vector = (int*) malloc(copyNum * sizeof(int));
		Ln_strategy = (int*) malloc(copyNum * (*iRows - 1) * sizeof(int));
		hipMalloc((void**)&d_Ln_vector, copyNum * sizeof(int));
		hipMalloc((void**)&d_Ln_strategy, copyNum * (*iRows - 1) * sizeof(int));
		hipMemcpy(d_mtx_to_vec, mtx_to_vec, *iRows * *iCols * sizeof(int), hipMemcpyHostToDevice);
		printf("num_ofBlock: %d, num_ofThread: %d\n",num_ofBlock,num_ofThread);
		L2<<<num_ofBlock,num_ofThread>>>(d_mtx_to_vec, steps, steps_remainder, d_Ln_vector, d_Ln_strategy, *iRows, *iCols);
		hipMemcpy(Ln_vector, d_Ln_vector, copyNum * sizeof(int), hipMemcpyDeviceToHost);
		hipMemcpy(Ln_strategy, d_Ln_strategy, copyNum * (*iRows - 1) * sizeof(int), hipMemcpyDeviceToHost);
		Ln_max = Ln_vector[0]; iMax = 0;
		for(i = 1; i < copyNum; i++){ if(Ln_max < Ln_vector[i]) {Ln_max = Ln_vector[i]; iMax = i;}}

		fp = fopen("strategy_L2.txt", "w");
		fprintf(fp,"1\n");
		for(i=0; i<(*iRows - 1); i++) {fprintf(fp, "%d\n", Ln_strategy[iMax * (*iRows - 1) + i]);}
		fclose(fp);
	}
	else if(*n == 3){ // if the order of the L norm is 3 then this part of the code will be executed.
		unsigned long long int *iNumPower, *d_iNumPower;

			for(i = 0; i < *iRows; i++){
				for(j = 0; j < *iCols; j++){
					mtx_to_vec[i * *iCols + j] = mtx[i][j];
				}
			}
		hipMalloc((void**)&d_mtx_to_vec, *iRows * *iCols * sizeof(int));
		Inner_num = pow(3, *iRows - 1);
		copyNum = NUM_OF_THREADS > Inner_num ? Inner_num : NUM_OF_THREADS; // The possible number of L norms can not be more than the number of threads
		num_ofThread = copyNum < devProp.warpSize ? copyNum : devProp.warpSize; // Number of threads in a block can not be bigger than the number of warps.
		num_ofBlock = copyNum/num_ofThread; copyNum = num_ofBlock * num_ofThread; //The number of blocks the code uses.
printf("Inner_num: %llu, copyNum: %llu, num_ofThread: %d, num_ofBlock: %d", Inner_num, copyNum, num_ofThread, num_ofBlock);
		maxRows = (int) (floor (NUM_OF_BITS / log2(*n)) + 1);
		if( *iRows > maxRows) {printf("Matrix is too big. The number of rows can not be more than %d.\n", maxRows); exit(-1);}
		if(*iCols > length) {printf("Matrix is too big. The length variable %d should be bigger or equal than %d.\n", length, *iCols); exit(-1);}
		printf("NUM_OF_BITS: %lu, maxRows: %d\n", NUM_OF_BITS ,maxRows);
		steps=Inner_num/copyNum; steps_remainder = Inner_num % copyNum;
		Ln_vector = (int*) malloc(copyNum * sizeof(int));
		Ln_strategy = (int*) malloc(copyNum * (*iRows - 1) * sizeof(int));
		iNumPower = (unsigned long long int*) malloc(maxRows * sizeof(unsigned long long int));
		hipMalloc((void**)&d_Ln_vector, copyNum * sizeof(int));
		hipMalloc((void**)&d_Ln_strategy, copyNum * (*iRows - 1) * sizeof(int));
		hipMalloc((void**)&d_iNumPower, (maxRows-1) * sizeof(unsigned long long int));
		iNumPower[0] = 1;
		for(i = 1; i < (maxRows-1); i++){iNumPower[i] = iNumPower[i-1] * 3; } // iNumPower is copied to the device memory to speed up the calculation of the ternary Gray code
		hipMemcpy(d_iNumPower, iNumPower, (maxRows-1) * sizeof(unsigned long long int), hipMemcpyHostToDevice);
		hipMemcpy(d_mtx_to_vec, mtx_to_vec, *iRows * *iCols * sizeof(int), hipMemcpyHostToDevice);
		printf("num_ofBlock: %d, num_ofThread: %d\n",num_ofBlock,num_ofThread);
		L3<<<num_ofBlock,num_ofThread>>>(d_mtx_to_vec, steps, steps_remainder, d_Ln_vector, d_Ln_strategy, *iRows, *iCols, d_iNumPower);
		hipMemcpy(Ln_vector, d_Ln_vector, copyNum * sizeof(int), hipMemcpyDeviceToHost);
		hipMemcpy(Ln_strategy, d_Ln_strategy, copyNum * (*iRows - 1) * sizeof(int), hipMemcpyDeviceToHost);
		Ln_max = Ln_vector[0]; iMax = 0;
		for(i = 1; i < copyNum; i++){if(Ln_max < Ln_vector[i]) {Ln_max = Ln_vector[i]; iMax = i; }}

		FILE *fp;
		fp = fopen("strategy_L3.txt", "w");
		for(i=0; i<(*iRows - 1); i++) {fprintf(fp, "%d\n", Ln_strategy[iMax * (*iRows - 1) + i]);}
		fprintf(fp,"0\n");
		fclose(fp);

		free(iNumPower);
		hipFree(d_iNumPower);
	}
	else{ // if the order of the L norm is bigger than 3, then this part of the code will be executed.
		int *iHelper, *d_iHelper;
		unsigned long long int *iNumPower, *d_iNumPower;

		for(i = 0; i < *iRows; i++){
			for(j = 0; j < *iCols; j++){
				mtx_to_vec[i * *iCols + j] = mtx[i][j];
			}
		}
		*n = ((*n < *iRows)? *n:*iRows);
		hipMalloc((void**)&d_mtx_to_vec, *iRows * *iCols * sizeof(int));
		Inner_num = pow(*n, *iRows - 1);
		copyNum = NUM_OF_THREADS > Inner_num ? Inner_num : NUM_OF_THREADS; // The possible number of L norms can not be more than the number of threads
		num_ofThread = copyNum < devProp.warpSize ? copyNum : devProp.warpSize; // Number of threads in a block can not be bigger than the number of warps.
		num_ofBlock = copyNum/num_ofThread; copyNum = num_ofBlock * num_ofThread; //The number of blocks the code uses.

		maxRows = (int) (floor (NUM_OF_BITS / log2(*n)) + 1);
		if(*iRows > maxRows) {printf("Matrix is too big. The number of rows can not be more than %d.\n", maxRows); exit(-1);}
		if(*iCols > length) {printf("Matrix is too big. The length variable %d should be bigger or equal than %d.\n", length, *iCols); exit(-1);}
		iHelper = (int*) calloc(2 * *n, sizeof(int));
		for(i = 0; i < *n; i++){
			iHelper[i] = i;
			iHelper[2 * *n-i-1]=i;
		}
		printf("NUM_OF_BITS: %lu, maxRows: %d\n", NUM_OF_BITS ,maxRows);
		steps=Inner_num/copyNum; steps_remainder = Inner_num % copyNum;
		Ln_vector = (int*) malloc(copyNum * sizeof(int));
		Ln_strategy = (int*) malloc(copyNum * (*iRows - 1) * sizeof(int));
		iNumPower = (unsigned long long int*) malloc(maxRows * sizeof(unsigned long long int));
		hipMalloc((void**)&d_Ln_vector, copyNum * sizeof(int));
		hipMalloc((void**)&d_Ln_strategy, copyNum * (*iRows - 1) * sizeof(int));
		hipMalloc((void**)&d_iNumPower, (maxRows-1) * sizeof(unsigned long long int));
		hipMalloc((void**)&d_iHelper, 2 * *n * sizeof(int));
		iNumPower[0] = 1;
		for(i = 1; i < (maxRows-1); i++){iNumPower[i] = iNumPower[i-1] * *n; } //printf("iNumPower: %llu\n", iNumPower[maxRows -2]);
		hipMemcpy(d_iNumPower, iNumPower, (maxRows-1) * sizeof(unsigned long long int), hipMemcpyHostToDevice);
		hipMemcpy(d_mtx_to_vec, mtx_to_vec, *iRows * *iCols * sizeof(int), hipMemcpyHostToDevice);
		hipMemcpy(d_iHelper, iHelper, 2 * *n * sizeof(int), hipMemcpyHostToDevice);
		printf("num_ofBlock: %d, num_ofThread: %d\n",num_ofBlock,num_ofThread); 
		Ln<<<num_ofBlock,num_ofThread>>>(d_mtx_to_vec, d_iHelper, steps, steps_remainder, d_Ln_vector, d_Ln_strategy, *iRows, *iCols, *n, d_iNumPower);
		hipMemcpy(Ln_vector, d_Ln_vector, copyNum * sizeof(int), hipMemcpyDeviceToHost);
		hipMemcpy(Ln_strategy, d_Ln_strategy, copyNum * (*iRows - 1) * sizeof(int), hipMemcpyDeviceToHost);
		Ln_max = Ln_vector[0]; iMax = 0;
		for(i = 1; i < copyNum; i++){if(Ln_max < Ln_vector[i]) {Ln_max = Ln_vector[i]; iMax = i; }}

		sprintf(fileOutput,"strategy_L%d.txt", *n);
		fp = fopen(fileOutput, "w");
		fprintf(fp, "#L%d is: %d\n", *n, Ln_max);
		for(i=0; i<(*iRows - 1); i++) {fprintf(fp, "%d\n", Ln_strategy[iMax * (*iRows - 1) + i]);}
		fprintf(fp,"0\n");
		fclose(fp);

		free(iNumPower);
		free(iHelper);
		hipFree(d_iNumPower);
		hipFree(d_iHelper);
	}
	
	printf("L%d is: %d\n", *n, Ln_max); // Write out the value of the L norm to the screen.
	
	free(Ln_vector); // Deallocates the vectors in the host memory.
	free(Ln_strategy);
	free(mtx_to_vec);
	
	hipFree(d_Ln_vector); // Deallocates the vectors in the device memory.
	hipFree(d_Ln_strategy);
	hipFree(d_mtx_to_vec);
}

void mtx_free(int* iRows, int** mtx){
	int i;
	for(i = 0; i < *iRows; i++){
		free(mtx[i]);
	}
	free(mtx);
}

int main(int argc, char *argv[]){
	char fileName[1024]; //The 'fileName' variable contains the name of the file.
	int iRows, iCols, **mtx, n; // These variables are the number of rows and columns of the matrix, the matrix itself, and 'n' is the order of the L norm one wants to calculate.
	fileN(fileName, argv, &argc); // The 'fileN' function ensures that the filename containing the matrix exists within the working directory.
	nNumber(&n, argv, &argc); // The 'nNumber' function ensures that the order of the L norm is calculatable by the code.
	mtx = mtx_read(&iRows, &iCols, fileName); // The 'mtx_read' function reads the file containing the matrix.
	calc_Lnorm(&n, &iRows, &iCols, mtx); // The 'calc_Lnorm' calculates the n order L norm of the matrix.
	mtx_free(&iRows, mtx); //The 'mtx_free' function deallocates the memory for the pointer mtx.
	return 0;     
}
