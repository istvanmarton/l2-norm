#include "hip/hip_runtime.h"
/*
compile:
  nvcc -gencode arch=compute_80,code=sm_80 -o L L.cu
use:
  ./L s7.mat
*/
#include<stdio.h>
#include<math.h>
#include<stdlib.h>

#define luint unsigned long long int
#define elemt int
#define maxHeight 96
#define maxWidth 96
#define maxWidthB 100
#define offs 32
#define procNum 1536

__device__ elemt best[1];
__device__ luint counter[1];
__device__ __constant__ elemt mat[maxHeight * maxWidthB];

__global__ void func(elemt *matt, const int grX, const int endX, const luint blocks) {
  elemt v1, v2, v3, tmp, m;
  luint b = blockIdx.x;
  const int l = threadIdx.x;
  elemt* mat = matt + l;
  elemt* matB = mat + offs;
  elemt* matC = matB + offs;
  elemt* cvec = matt + maxWidth;
  int gr   = grX  * maxWidthB;
  int end  = endX * maxWidthB;
  int mtx;

beg:
  mtx = 0;
  m = best[0];
  if (mtx == end) goto end;

  v1 = mat[mtx];
  v2 = matB[mtx];
  v3 = matC[mtx];
  while (mtx < gr - maxWidthB) {
    mtx += maxWidthB;
    tmp = mat[mtx];
    if (b & 1) tmp = -tmp;
    v1 += tmp;
    tmp = matB[mtx];
    if (b & 1) tmp = -tmp;
    v2 += tmp;
    tmp = matC[mtx];
    if (b & 1) tmp = -tmp;
    v3 += tmp;
    b >>= 1;
  }

d:
  mtx += maxWidthB;
  tmp = __reduce_add_sync(0xffffffff, abs(v1) + abs(v2) + abs(v3));

  if (mtx == end) {
    m = max(m, tmp);
    goto u;
  }
  if (tmp + cvec[mtx] > m) {
    b <<= 1;
    v1 += mat[mtx];
    v2 += matB[mtx];
    v3 += matC[mtx];
    goto d;
  }

u:
  if (mtx == gr) goto end;
  mtx -= maxWidthB;
  if (b & 1) {
    b >>= 1;
    v1 += mat[mtx];
    v2 += matB[mtx];
    v3 += matC[mtx];
    goto u;
  }
  b++;
  v1 -= mat[mtx] << 1;
  v2 -= matB[mtx] << 1;
  v3 -= matC[mtx] << 1;
  goto d;

end:
  if (l == 0) {
    tmp = best[0];
    if (m > tmp) atomicMax(best, m);
    b = atomicAdd(counter, (luint)1);
  }
  b = __shfl_sync(0xffffffff, b, 0);
  if ((long long int)b < blocks) goto beg;
}

elemt comp(int ofs, int granM, int height, elemt guess) {
  elemt *mtx;
  hipGetSymbolAddress((void**)&mtx, mat);
  int gran = min(max(11, granM) + 1, height) - 1;
  luint blocks = (luint)1 << gran;
//  luint blk = (luint)1 << min(gran, 11);
  luint blk = gran < 32 ? min((int)blocks, procNum) : procNum;
  hipMemcpyToSymbol(HIP_SYMBOL(best), &guess, sizeof(elemt));
  hipMemcpyToSymbol(HIP_SYMBOL(counter), &blk, sizeof(luint));
  func<<<blk, 32>>>(mtx + ofs * maxWidthB, gran + 1, height, blocks);
  hipMemcpyFromSymbol(&guess, HIP_SYMBOL(best), sizeof(elemt));
  hipMemcpy(mtx + ofs * maxWidthB + maxWidth, &guess, sizeof(elemt), hipMemcpyHostToDevice);
  return guess;
}

int main(int argc, char* argv[]){
  elemt mtx[maxHeight * maxWidthB];
  FILE* fp = fopen(argv[1], "r");
  char *line = NULL;
  size_t len = 0;
  int height = 0, width = 0;
  while (1) {
    ssize_t read = getline(&line, &len, fp);
    if (read <= 1) break;
    int offset = 0, inc, j = 0;
    while (1) {
      int res = sscanf(line + offset, "%d%n", &mtx[height * maxWidthB + j], &inc);
      if (res < 1) break;
      offset += inc;
      j++;
    }
    for (int k = j; k < maxWidth; k++) mtx[height * maxWidthB + k] = 0;
    mtx[height * maxWidthB + maxWidth] = 1 << 30;
    height++;
    width = max(width, j);
  }
  free(line);
  fclose(fp);
  printf("%d x %d matrix\n", height, width);

  hipMemcpyToSymbol(HIP_SYMBOL(mat), mtx, maxWidthB * height * sizeof(elemt));

  elemt l1;
  int lim = height / 2 + 2;
  for (int i = height; i >= 0; i--) {
    if (i > lim || i == 0) {
      l1 = comp(i, (height - i) / 2 + 2, height - i, l1);
//      printf("L1 norm at %d: %d\n", i, l1);
      }
  }
  printf("L norm: %d\n", l1);

  return 0;
}
