#include "hip/hip_runtime.h"
/*
compile:
  nvcc -gencode arch=compute_80,code=sm_80 -o L2 L2.cu
use:
  ./L2 s6.mat
*/
#include<stdio.h>
#include<math.h>
#include<stdlib.h>

#define luint unsigned long long int
#define elemt int
#define maxHeight 96
#define maxWidth 96
#define maxWidthB 100
#define offs 32
#define procNum 1536

__device__ elemt best[1];
__device__ luint counter[1];
__device__ __constant__ elemt mat[maxHeight * maxWidthB];

__global__ void func(elemt *matt, const int grX, const int endX, const luint blocks) {
  elemt v1, v2, v3, w1, w2, w3, tmpv, tmpw, m;
//  bool bv, bw;
  luint b = blockIdx.x;
  const int l = threadIdx.x;
  elemt* mat = matt + l;
  elemt* matB = mat + offs;
  elemt* matC = matB + offs;
  elemt* cvec = matt + maxWidth;
  int gr   = grX  * maxWidthB;
  int end  = endX * maxWidthB;
  int mtx;

beg:
  mtx = 0;
  m = best[0];
  if (mtx == end) goto end;
//  bv = bw = true;

  v1 = mat[mtx];
  v2 = matB[mtx];
  v3 = matC[mtx];
  w1 = w2 = w3 = 0;
  while (mtx < gr - maxWidthB) {
    mtx += maxWidthB;
    if (b & 1) {
      v1 += mat[mtx];
      v2 += matB[mtx];
      v3 += matC[mtx];
    } else {
      w1 += mat[mtx];
      w2 += matB[mtx];
      w3 += matC[mtx];
    }
    b >>= 1;
  }

d:
  mtx += maxWidthB;
  /*if (bv) {bv = false;*/ tmpv = __reduce_add_sync(0xffffffff, abs(v1) + abs(v2) + abs(v3));//}
  /*if (bw) {bw = false;*/ tmpw = __reduce_add_sync(0xffffffff, abs(w1) + abs(w2) + abs(w3));//}

  if (mtx == end) {
    m = max(m, tmpv + tmpw);
    goto u;
  }
  if (tmpv + tmpw + cvec[mtx] > m) {
    b <<= 1;
//    bv = true;
    v1 += mat[mtx];
    v2 += matB[mtx];
    v3 += matC[mtx];
    goto d;
  }

u:
  if (mtx == gr) goto end;
  mtx -= maxWidthB;
  if (b & 1) {
    b >>= 1;
//    bw = true;
    w1 -= mat[mtx];
    w2 -= matB[mtx];
    w3 -= matC[mtx];
    goto u;
  }
  b++;
//  bv = bw = true;
  v1 -= mat[mtx];
  w1 += mat[mtx];
  v2 -= matB[mtx];
  w2 += matB[mtx];
  v3 -= matC[mtx];
  w3 += matC[mtx];
  goto d;

end:
  if (l == 0) {
    tmpv = best[0];
    if (m > tmpv) atomicMax(best, m);
    b = atomicAdd(counter, (luint)1);
  }
  b = __shfl_sync(0xffffffff, b, 0);
  if ((long long int)b < blocks) goto beg;
}

elemt comp(int ofs, int granM, int height, elemt guess) {
  elemt *mtx;
  hipGetSymbolAddress((void**)&mtx, mat);
  int gran = min(max(11, granM) + 1, height) - 1;
  luint blocks = (luint)1 << gran;
//  luint blk = (luint)1 << min(gran, 11);
  luint blk = gran < 32 ? min((int)blocks, procNum) : procNum;
  hipMemcpyToSymbol(HIP_SYMBOL(best), &guess, sizeof(elemt));
  hipMemcpyToSymbol(HIP_SYMBOL(counter), &blk, sizeof(luint));
  func<<<blk, 32>>>(mtx + ofs * maxWidthB, gran + 1, height, blocks);
  hipMemcpyFromSymbol(&guess, HIP_SYMBOL(best), sizeof(elemt));
  hipMemcpy(mtx + ofs * maxWidthB + maxWidth, &guess, sizeof(elemt), hipMemcpyHostToDevice);
  return guess;
}

int main(int argc, char* argv[]){
  elemt mtx[maxHeight * maxWidthB];
  FILE* fp = fopen(argv[1], "r");
  char *line = NULL;
  size_t len = 0;
  int height = 0, width = 0;
  while (1) {
    ssize_t read = getline(&line, &len, fp);
    if (read <= 1) break;
    int offset = 0, inc, j = 0;
    while (1) {
      int res = sscanf(line + offset, "%d%n", &mtx[height * maxWidthB + j], &inc);
      if (res < 1) break;
      offset += inc;
      j++;
    }
    for (int k = j; k < maxWidth; k++) mtx[height * maxWidthB + k] = 0;
    mtx[height * maxWidthB + maxWidth] = 1 << 30;
    height++;
    width = max(width, j);
  }
  free(line);
  fclose(fp);
  printf("%d x %d matrix\n", height, width);

  hipMemcpyToSymbol(HIP_SYMBOL(mat), mtx, maxWidthB * height * sizeof(elemt));

  elemt l1;
  int lim = height / 2 + 2;
  for (int i = height; i >= 0; i--) {
    if (i > lim || i == 0) {
      l1 = comp(i, (height - i) / 2 + 2, height - i, 0);
//      printf("L1 norm at %d: %d\n", i, l1);
      }
  }
  printf("L2 norm: %d\n", l1);

  return 0;
}
